#include "hip/hip_runtime.h"
#include "simulation.cuh"
#include <stdio.h>

__host__ void initialize_doses(double* doses, int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            doses[i * N + j] = 0;
        }
    }
    return;
}

__host__ void initialize_densities_random(double* densities, int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            densities[i * N + j] = uniform_dist(random_engine);
        }
    }
    return;
}

__host__ void initialize_densities_constant(double* densities, int N, double density)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            densities[i * N + j] = density;
        }
    }
    return;
}

__host__ void initialize_densities_centered_gaussian(double* densities, int N, double max_density, double spread)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            int mid = N / 2; // middle pixel
            int x = i - mid;
            int y = j - mid;
            double std_dev = spread * N;
            densities[i * N + j] = max_density * exp(-(x * x + y * y) / (2 * std_dev * std_dev));
        }
    }
    return;
}

__host__ void
initialize_densities_random_gaussians(double* densities, int N, int n_gaussians, double max_density, double spread)
{
    double std_dev = spread * N;
    double highest = 0;

    // Add the Gaussians
    for (int k = 0; k < n_gaussians; k++)
    {
        int mid_x = floor(uniform_dist(random_engine) * N);
        int mid_y = floor(uniform_dist(random_engine) * N);
        for (int i = 0; i < N; i++)
        {
            for (int j = 0; j < N; j++)
            {
                int x = i - mid_x;
                int y = j - mid_y;
                densities[i * N + j] += max_density * exp(-(x * x + y * y) / (2 * std_dev * std_dev));
                highest = fmax(highest, densities[i * N + j]);
            }
        }
    }

    // Normalize the resulting density distribution
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            densities[i * N + j] = max_density * densities[i * N + j] / highest;
        }
    }
    return;
}

__host__ void extend_ray_groups(RegionGroup *region_groups, int rays_per_group, int region_index, int max_index){
    RayGroup *ray_groups = region_groups[region_index].my_ray_groups;
    ray_groups = (RayGroup *) realloc(ray_groups, (max_index + 0)* 2 * sizeof(RayGroup));
    region_groups[region_index].my_ray_groups = ray_groups;
    for (int i = max_index; i < max_index * 2; i++){
        region_groups[region_index].my_ray_groups[i].my_rays = (Ray *) malloc(rays_per_group * sizeof(Ray));
        region_groups[region_index].my_ray_groups[i].my_size = 0;
        region_groups[region_index].my_ray_groups[i].max_size = rays_per_group;
    }
    region_groups[region_index].max_size = max_index * 2;
    
}

__host__ void initialize_region_groups(RegionGroup *region_groups, int num_regions, int ray_groups_per_region, int rays_per_group){
    for (int i = 0; i < num_regions; i++){
        region_groups[i].my_ray_groups = (RayGroup *) malloc(ray_groups_per_region * sizeof(RayGroup));
        region_groups[i].my_size = -1;
        region_groups[i].max_size = ray_groups_per_region;
        for (int j = 0; j < ray_groups_per_region; j++){
            Ray *r = (Ray *) malloc(rays_per_group * sizeof(Ray));
            region_groups[i].my_ray_groups[j].my_rays = r;
            region_groups[i].my_ray_groups[j].my_size = 0;
            region_groups[i].my_ray_groups[j].max_size = rays_per_group;
        }
    }
}

__host__ void write_to_csv_file(double* grid_data, int N, const std::string& filename)
{
    std::ofstream output;
    output.open(filename);
    for (int j = 0; j < N; j++)
    {
        for (int i = 0; i < N - 1; i++)
        {
            output << grid_data[i * N + j] << ",";
        }
        output << grid_data[N - 1 + j] << "\n";
    }
    output.close();
    return;
}

__device__ void init_curand_state(hiprandState_t* state)
{
    // Initialize random kernel
    int tId = threadIdx.x + (blockIdx.x * blockDim.x);
    hiprand_init((unsigned long long)clock(), tId, 0, state);
    //printf("myID: %d\n", tId);
    return;
}

__device__ double uniform_angle_dist(hiprandState_t* state) { return 2 * M_PI * hiprand_uniform_double(state); }

__device__ double normal_angle_dist(hiprandState_t* state, double mean, double std_dev)
{
    return mean + std_dev * hiprand_normal_double(state);
}

__device__ double random_source_angle_uniform()
{
    hiprandState state;
    init_curand_state(&state);

    double angle = uniform_angle_dist(&state);
    
    // Normalize angle
    if (abs(angle - 2 * M_PI) < PARAM_EPSILON)// if angle == 2pi, angle = 0
    {
        angle = 0;
    }
    return angle;
}

__host__ double random_source_angle_normal()
{
    double angle = normal_dist(random_engine);
    // Normalize angle
    while (angle < 0.0)
    {
        angle += 2 * M_PI;
    }
    while (angle >= 2 * M_PI)
    {
        angle -= 2 * M_PI;
    }
    return angle;

}


__device__ bool out_of_bounds(Pixel current_pixel, int N)
{
    return (current_pixel.first < 0 || current_pixel.first >= N || current_pixel.second < 0 ||
            current_pixel.second >= N);
}

__host__ void spawn_primary_rays(RegionGroup *region_groups, int num_primary_rays, int N, int M)
{
    for (int i = 0; i < num_primary_rays; i++){

        // Randomly select source angle from normal distribution
        double source_angle = random_source_angle_normal();
        // Calculate initial ray position
        double horiz_dist_from_center = PARAM_D * N * tan(source_angle); // horizontal distance from center of top edge
        int middle_pixel = N / 2;
        double horiz_dist_from_left = middle_pixel + horiz_dist_from_center;
        
        // If ray does not miss grid entirely, spawn it
        if (horiz_dist_from_left < 0 || horiz_dist_from_left >= N ||
            (source_angle >= M_PI / 2 && source_angle <= 3 * M_PI / 2))
        {
            continue;
            // DEBUG(DB_INIT_PRI, std::cout << "New primary ray missed the grid, not adding" << std::endl);
        }
        
        double horiz_dist_from_left_rounded = floor(horiz_dist_from_left);
        Pixel spawn_pixel;
        spawn_pixel.first = horiz_dist_from_left_rounded;
        spawn_pixel.second = 0; // always starts from top of grid
        double edge_dist = horiz_dist_from_left - horiz_dist_from_left_rounded;

        Region region = get_region(spawn_pixel, N, M);
        Ray r = Ray::primary(source_angle, spawn_pixel, PIXEL_EDGE::TOP, edge_dist, region);
        
        int Rx = floor((float) N / M);
        int region_index = region.second * Rx + region.first;
        int max_index = region_groups[region_index].max_size;
        int group_index = region_groups[region_index].my_size + 1;
        

        if (group_index > max_index - 1){
            //printf("resizing array of ray groups\n");
            //printf("max_index: %d\t group_index: %d\n", max_index, group_index);
            //printf("Group: %d\t Region: %d, %d\n\n", region_index, region.first, region.second);
            extend_ray_groups(region_groups, PARAM_KS + 1, region_index, max_index);
            //printf("finished resizing\n");
        }
        //printf("max_index: %d\t group_index: %d\n", region_groups[region_index].max_size, group_index);
        //printf("Group: %d\t Region: %d, %d\n\n", region_index, region.first, region.second);
        region_groups[region_index].my_ray_groups[group_index].my_rays[0] = r; // add new ray to the beginning of ray group
        region_groups[region_index].my_ray_groups[group_index].my_size = 1; // update size
        region_groups[region_index].my_size = group_index;
        //printf("Angle: %.2f\tPosition: %d, %d\n", source_angle, spawn_pixel.first, spawn_pixel.second);
        //printf("max_index: %d\t group_index: %d\n", max_index, group_index);
        //printf("Group: %d\t Region: %d, %d\n\n", region_index, region.first, region.second);
    }
    
    return;
}

__device__ void spawn_secondary_rays(RayGroup* group, Pixel spawn_pixel, double total_energy, int N)
{
    // DEBUG(DB_INIT_SEC, std::cout << "Spawning " << PARAM_KS << " secondary rays from pixel " << spawn_pixel.first <<
    // ","
    //                                 << spawn_pixel.second << "..." << std::endl);
    for (int i = 0; i < PARAM_KS; i++)
    {
        double source_angle = random_source_angle_uniform(); // uniform random source angle
        double partial_energy = total_energy / PARAM_KS;
        Ray new_ray = Ray::secondary_from_center(source_angle, spawn_pixel, partial_energy);
        Pixel current_pixel = new_ray.get_current_pixel();
        if (out_of_bounds(current_pixel, N))
        {
            // DEBUG(DB_INIT_SEC, std::cout << "Ray is out of bounds, not adding" << std::endl);
        }
        else
        {
            group->my_rays[group->my_size] = new_ray;
            group->my_size++;

            // DEBUG(DB_INIT_SEC, std::cout << "Ray is in bounds, adding" << std::endl);
        }
    }
    // DEBUG(DB_INIT_SEC, std::cout << "Done" << std::endl << std::endl);
    return;
}

__device__ bool random_interact(Pixel target_pixel, double distance, double* densities, int N)
{
    int i = target_pixel.first, j = target_pixel.second;
    double density = densities[i * N + j];
    double l_ep = density * distance; // effective path length travelled in pixel
    double probability = std::exp(-PARAM_A / l_ep);

    hiprandState state;
    init_curand_state(&state);
    double rand = hiprand_uniform_double(&state);
    return (rand < probability);
}

__device__ void
transfer_energy(Ray* ray, Pixel target_pixel, double unscaled_energy, double* densities, double* doses, int N)
{
    int i = target_pixel.first, j = target_pixel.second;
    double density = densities[i * N + j];
    double energy_to_transfer = unscaled_energy * density; // scale energy by pixel density
    double current_ray_energy = ray->get_current_energy();

    // Ray cannot transfer more energy that it has
    energy_to_transfer = fmin(energy_to_transfer, current_ray_energy);

    // Remove energy from ray and add it to pixel dose
    ray->set_current_energy(current_ray_energy - energy_to_transfer);
    //printf("Attempting to transfer %0.6f energy\n", energy_to_transfer);
    //printf("Pixel energy before transfer: %0.6f\n", doses[i * N + j]);
    doses[i * N + j] += energy_to_transfer;
    //printf("Pixel energy after transfer: %0.6f\n", doses[i * N + j]);
    return;
}

__device__ int evolve_rays(RayGroup* group, double* densities, double* doses, int N)
{
    int rays_evolved = 0;

    for (int i = 0; i < group->my_size; i++)
    {
        Ray* r = &group->my_rays[i];
        // Only evolve active rays
        if (r->is_active())
        {
            // Trace ray
            // DEBUG(DB_TRACE, std::cout << "Tracing ray " << i << std::endl);
            TraceHistory rtrace = r->trace();
            Pixel visited_pixel = rtrace.visited;
            double travel_distance = rtrace.distance; // distance traveled in visited pixel
            rays_evolved++;

            if (r->is_primary()) // primary ray
            {
                // DEBUG(DBevolve_PRI, std::cout << "Primary ray " << i << "  visited pixel " << visited_pixel.first
                //                                               << "," << visited_pixel.second << " with travel dist "
                //                                               << travel_distance
                //                                               << std::endl);
                if (random_interact(visited_pixel, travel_distance, densities, N))
                {
                    printf("block %d, thread %d primary ray interacted at pixel %d,%d\n", blockIdx.x, threadIdx.x,
                           visited_pixel.first, visited_pixel.second);
                    // DEBUG(DBevolve_PRI, std::cout << "Primary ray " << i << " interacted" << std::endl);
                    // Deposit energy to pixel
                    // DEBUG(DBevolve_PRI, std::cout << "Depositing energy to pixel" << std::endl);
                    // DEBUG(DBevolve_PRI, std::cout << "Starting energy " << r->get_current_energy() << std::endl);
                    printf("distance traveled %0.6f\n", travel_distance);
                    double energy_to_deposit = PARAM_F * travel_distance * r->get_current_energy();
                    transfer_energy(r, visited_pixel, energy_to_deposit, densities, doses, N);
                    // DEBUG(DBevolve_PRI, std::cout << "Energy after deposit " << r->get_current_energy() <<
                    // std::endl); DEBUG(DBevolve_PRI, std::cout << "Will spawn secondary rays next" << std::endl <<
                    // std::endl); Spawn secondary rays, transferring remaining energy to them
                    spawn_secondary_rays(group, visited_pixel, r->get_current_energy(), N);
                    r->set_current_energy(0);
                }
                else
                {
                    // DEBUG(DBevolve_PRI, std::cout << "No interaction" << std::endl << std::endl);
                }
            }
            else // secondary ray
            {
                // DEBUG(DBevolve_SEC, std::cout << "Secondary ray " << i << " visited pixel " << visited_pixel.first
                //                                               << "," << visited_pixel.second << " with travel dist "
                //                                               << travel_distance
                //                                               << std::endl);
                double energy_to_deposit = PARAM_G * travel_distance;
                // DEBUG(DBevolve_SEC, std::cout << "Depositing energy to pixel" << std::endl);
                // DEBUG(DBevolve_SEC, std::cout << "Starting energy " << r->get_current_energy() << std::endl);
                // DEBUG(DBevolve_SEC, std::cout << "Unscaled energy to deposit " << energy_to_deposit << std::endl);
                transfer_energy(r, visited_pixel, energy_to_deposit, densities, doses, N);
                // DEBUG(DBevolve_SEC, std::cout << "Energy after deposit " << r->get_current_energy() << std::endl
                //                                               << std::endl);
            }

            // Deactivate ray if out of energy or outside of the grid bounds
            if (r->get_current_energy() < PARAM_MINERGY || out_of_bounds(r->get_current_pixel(), N))
            {
                // DEBUG(DBevolve_SEC, std::cout << "Ray " << i << " is out of energy or bounds, deactivating"
                //                                               << std::endl
                //                                               << std::endl);
                r->deactivate();
            }
        }
    }
    return rays_evolved;
}

__device__ void evolve_to_completion(RayGroup* group, double* densities, double* doses, int N)
{
    int rays_evolved = group->my_size;
    //  int raysevolved = ray_group->size();
    while (rays_evolved > 0)
    {
        rays_evolved = evolve_rays(group, densities, doses, N);
        // DEBUG(DB_GENERAL, std::cout << raysevolved << " rays evolved" << std::endl);
        // DEBUG(DB_GENERAL, std::cout << (ray_group->size() - prev_num_rays) << " rays added" << std::endl <<
        // std::endl)
    }

    return;
}
__device__ void run_serial(RegionGroup *region_groups, double* densities, double* doses, int N, int M)
{
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
    // Each primary ray is done serially as its own individual ray group
    

    /// check to see if GPU received all the correct data. It does
    
    int num_regions = pow(ceilf(N / M), 2); // number of regions
    printf("N: %d\tM: %d\tnum_regions: %d\n", N, M, num_regions);
    int count = 0;
    for (int i = 0; i < num_regions; i++){
        RegionGroup cur_region_group = region_groups[i];
        int cur_region_group_size = cur_region_group.my_size;
        //printf("cur_region_group_size: %d\n", cur_region_group_size);
        for (int j = 0; j < cur_region_group_size; j++){
            RayGroup cur_ray_group = cur_region_group.my_ray_groups[j];
                Ray r = cur_ray_group.my_rays[0];
                Pixel position = r.get_current_pixel();
                Region region = r.get_current_region();
                printf("Primary %d: G: %d, P: %d, %d\tR: %d, %d\n", count, i, position.first, position.second, region.first, region.second);
                count++;
        }
    }
    return;
    
    /*
    for (int i = 0; i < num_primary_rays; i++)
    {
        printf("Hello from block %d, thread %d. I'm running primary ray %d\n", blockIdx.x, threadIdx.x, i);
        // Just running one primary ray (and its secondaries) at a time for now
        // TODO: might want to change how memory allocation/reallocation is done
        RayGroup primary_ray_group;
        int max_num_rays = 1 + PARAM_KS; // at most one primary ray plus all secondaries
        Ray* rays;
        rays = (Ray*)malloc(max_num_rays * sizeof(Ray));
        primary_ray_group.my_rays = rays;
        primary_ray_group.my_size = 0;

        // DEBUG(DB_INIT_PRI, std::cout << "Spawning " << num_primary_rays << " primary rays..." << std::endl);
        //spawn_primary_ray(&primary_ray_group, N);
        // DEBUG(DB_INIT_PRI, std::cout << "Done, " << ray_group.size() << " rays added" << std::endl << std::endl);
        // DEBUG(DB_GENERAL, std::cout << "Evolving rays..." << std::endl)
        evolve_to_completion(&primary_ray_group, densities, doses, N);
        // DEBUG(DB_GENERAL, std::cout << "Done" << std::endl);
        free(primary_ray_group.my_rays);
    }
    return;
    */
}

/// Kernel function for base-GPU
__global__ void run_rays(RegionGroup *region_groups, double* densities, double* doses, int N, int M)
{
    run_serial(region_groups, densities, doses, N, M);
}

// TODO could actually make N a command line argument, right?
int main(void)
{
    DEBUG(DB_GPU, std::cout << "Starting simulation, allocating grids" << std::endl);
    int N = 1000; // grid size in pixels per side
    int M = 250; // region size in pixels per side
    int num_primary_rays = 100; // number of primary rays to be scattered across the grid
    int rays_per_group = PARAM_KS + 1; // initial capacity of a ray group
    int ray_groups_per_region = 10; // intial capacity of a region group

    // Storing the N by N grid data as 1D arrays of length N*N
    // such that element i,j is at index i * N + j
    // Currently for the model I think i corresponds to x and j corresponds to y
    double *densities, *doses;
    hipMallocManaged(&densities, N * N * sizeof(double));
    hipMallocManaged(&doses, N * N * sizeof(double));

    DEBUG(DB_GPU, std::cout << "Initializing densities" << std::endl);
    // initialize_densities_random(densities, N);
    initialize_densities_constant(densities, N, 0.3);

    DEBUG(DB_GPU, std::cout << "Initializing doses to zero" << std::endl);
    initialize_doses(doses, N);
    size_t heap_limit = pow(2, 26); // default 8MB, this sets to 64 MB
    hipDeviceSetLimit(hipLimitMallocHeapSize, heap_limit);
 
    DEBUG(DB_GPU, std::cout << "Writing densities" << std::endl);
    //write_to_csv_file(densities, N, "../../plot/densities.csv");
    write_to_csv_file(densities, N, "densities.csv");
    


    DEBUG(DB_GPU, std::cout << "Running rays on threads" << std::endl);
    int num_regions = pow(ceil(N / M), 2); // number of regions
    DEBUG(DB_GPU, std::cout << "Number of regions: " << num_regions << std::endl);
    RegionGroup *region_groups = (RegionGroup *) malloc(num_regions * sizeof(RegionGroup));
    //RayGroup *groups = (RayGroup *) malloc(num_regions * sizeof(RayGroup)); // region groups
    initialize_region_groups(region_groups, num_regions, ray_groups_per_region, rays_per_group);
    //Ray *rays = (Ray *) malloc(num_primary_rays * sizeof(Ray));
    spawn_primary_rays(region_groups, num_primary_rays, N, M);
    
    int count = 0;
    for (int i = 0; i < num_regions; i++){
        RegionGroup cur_region_group = region_groups[i];
        int cur_region_group_size = cur_region_group.my_size;
        for (int j = 0; j < cur_region_group_size; j++){
            RayGroup cur_ray_group = cur_region_group.my_ray_groups[j];
                Ray r = cur_ray_group.my_rays[0];
                Pixel position = r.get_current_pixel();
                Region region = r.get_current_region();
                printf("Primary %d: G: %d, P: %d, %d\tR: %d, %d\n", count, i, position.first, position.second, region.first, region.second);
                count++;
        }
    }
    RegionGroup *region_groups_cuda;
    
    hipMallocManaged(&region_groups_cuda, num_regions * sizeof(RegionGroup));
    hipMemcpy(region_groups_cuda, region_groups, num_regions * sizeof(RegionGroup), hipMemcpyHostToDevice);
    for (int i = 0; i < num_regions; i++){
        ray_groups_per_region = region_groups[i].my_size;
        RayGroup *ray_groups_cuda;
        hipMallocManaged(&ray_groups_cuda, ray_groups_per_region * sizeof(RayGroup));

        hipMemcpy(ray_groups_cuda, region_groups[i].my_ray_groups, 
            ray_groups_per_region * sizeof(RayGroup), hipMemcpyHostToDevice);

        region_groups_cuda[i].my_ray_groups = ray_groups_cuda;
        region_groups_cuda[i].my_size = region_groups[i].my_size;
        region_groups_cuda[i].max_size = region_groups[i].max_size;

        for (int j = 0; j < ray_groups_per_region; j++){
            Ray *rays_cuda;
            hipMallocManaged(&rays_cuda, rays_per_group * sizeof(Ray));

            hipMemcpy(rays_cuda, region_groups[i].my_ray_groups[j].my_rays,
                rays_per_group * sizeof(Ray), hipMemcpyHostToDevice);
            region_groups_cuda[i].my_ray_groups[j].my_rays = rays_cuda;
            region_groups_cuda[i].my_ray_groups[j].my_size = region_groups[i].my_ray_groups[j].my_size;
            region_groups_cuda[i].my_ray_groups[j].max_size = region_groups[i].my_ray_groups[j].max_size;
        }
    }
    

    int grid_size = 1; // number of thread blocks
    int block_size = 1;   // TODO 1 thread per block, does this make sense?
    run_rays<<<grid_size, block_size>>>(region_groups_cuda, densities, doses, N, M);

    // Wait for GPU computation to finish
    //hipDeviceSynchronize();

    DEBUG(DB_GPU, std::cout << "Writing doses" << std::endl);
    //write_to_csv_file(doses, N, "../../plot/doses.csv");
    write_to_csv_file(doses, N, "doses.csv");
    DEBUG(DB_GPU, std::cout << "I'm the main function, look at me!" << std::endl);

    hipFree(densities);
    hipFree(doses);

    return 0;
}

Region get_region(Pixel position, int N, int M){
    int px = position.first;
    int py = position.second;

    Region region;
    region.first = floor((float) px / M);
    region.second = floor((float) py / M);
    return region;
}

