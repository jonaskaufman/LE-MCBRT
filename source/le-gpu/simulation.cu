#include "hip/hip_runtime.h"
#include "simulation.cuh"
#include <stdio.h>

////////// GRID INITIALIZATION //////////

__host__ void initialize_doses(double* doses, int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            doses[i + j * N] = 0;
        }
    }
    return;
}

__host__ void initialize_densities_random(double* densities, int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            densities[i + j * N] = uniform_dist(random_engine);
        }
    }
    return;
}

__host__ void initialize_densities_constant(double* densities, int N, double density)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            densities[i + j * N] = density;
        }
    }
    return;
}

__host__ void initialize_densities_centered_gaussian(double* densities, int N, double max_density, double spread)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            int mid = N / 2; // middle pixel
            int x = i - mid;
            int y = j - mid;
            double std_dev = spread * N;
            densities[i + j * N] = max_density * exp(-(x * x + y * y) / (2 * std_dev * std_dev));
        }
    }
    return;
}

__host__ void
initialize_densities_random_gaussians(double* densities, int N, int n_gaussians, double max_density, double spread)
{
    double std_dev = spread * N;
    double highest = 0;

    // Add the Gaussians
    for (int k = 0; k < n_gaussians; k++)
    {
        int mid_x = floor(uniform_dist(random_engine) * N);
        int mid_y = floor(uniform_dist(random_engine) * N);
        for (int i = 0; i < N; i++)
        {
            for (int j = 0; j < N; j++)
            {
                int x = i - mid_x;
                int y = j - mid_y;
                densities[i + j * N] += max_density * exp(-(x * x + y * y) / (2 * std_dev * std_dev));
                highest = fmax(highest, densities[i * N + j]);
            }
        }
    }

    // Normalize the resulting density distribution
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            densities[i + j * N] = max_density * densities[i * N + j] / highest;
        }
    }
    return;
}

////////// OUTPUT //////////

__host__ void write_to_csv_file(double* grid_data, int N, const std::string& filename)
{
    std::ofstream output;
    output.open(filename);
    for (int j = 0; j < N; j++)
    {
        for (int i = 0; i < N - 1; i++)
        {
            output << grid_data[i + j * N] << ",";
        }
        output << grid_data[N - 1 + j * N] << "\n";
    }
    output.close();
    return;
}

////////// RANDOMIZATION //////////

__device__ void init_curand_state(hiprandState_t* state)
{
    // Initialize random kernel
    int tId = threadIdx.x + (blockIdx.x * blockDim.x);
    hiprand_init((unsigned long long)clock(), tId, 0, state);
    return;
}

__device__ double uniform_angle_dist(hiprandState_t* state) { return 2 * M_PI * hiprand_uniform_double(state); }

__device__ double normal_angle_dist(hiprandState_t* state, double mean, double std_dev)
{
    return mean + std_dev * hiprand_normal_double(state);
}

__device__ double random_source_angle_uniform()
{
    hiprandState state;
    init_curand_state(&state);

    double angle = uniform_angle_dist(&state);

    // Angle of 2 pi goes to 0
    if (abs(angle - 2 * M_PI) < PARAM_EPSILON)
    {
        angle = 0.0;
    }
    return angle;
}

__host__ double random_source_angle_normal()
{
    double angle = normal_dist(random_engine);

    // Normalize angle
    while (angle < 0.0)
    {
        angle += 2 * M_PI;
    }
    while (angle >= 2 * M_PI)
    {
        angle -= 2 * M_PI;
    }
    return angle;
}

////////// RAY LOCATION CHECKING //////////

__host__ __device__ Region get_region(Pixel position, int N, int M)
{
    int px = position.first;
    int py = position.second;
    Region region;
    region.first = px / M;
    region.second = py / M;
    return region;
}

__host__ __device__ int get_region_index(Pixel position, int N, int M)
{
    Region region = get_region(position, N, M);
    int L = N / M; // number of regions per side
    return region.first + region.second * L;
}

__device__ bool out_of_bounds(Pixel current_pixel, int N)
{
    return (current_pixel.first < 0 || current_pixel.first >= N || current_pixel.second < 0 ||
            current_pixel.second >= N);
}

////////// RAY CREATION //////////

__host__ void spawn_primary_rays(
    std::vector<RegionGroup>& region_groups, int num_primary_rays, int max_rays_per_ray_group, int N, int M)
{
    int L = N / M; // number of regions per side
    for (int i = 0; i < num_primary_rays; i++)
    {
        // Randomly select source angle from normal distribution
        double source_angle = random_source_angle_normal();

        // Calculate initial ray position
        double horiz_dist_from_center = PARAM_D * N * tan(source_angle); // horizontal distance from center of top edge
        int middle_pixel = N / 2;
        double horiz_dist_from_left = middle_pixel + horiz_dist_from_center;

        // Check if ray missed the grid entirely
        if (horiz_dist_from_left < 0 || horiz_dist_from_left >= N ||
            (source_angle >= M_PI / 2 && source_angle <= 3 * M_PI / 2))
        {
            continue;
        }

        // If not, spawn it
        double horiz_dist_from_left_rounded = floor(horiz_dist_from_left);
        Pixel spawn_pixel;
        spawn_pixel.first = horiz_dist_from_left_rounded;
        spawn_pixel.second = 0; // always starts from top of grid
        double edge_dist = horiz_dist_from_left - horiz_dist_from_left_rounded;
        Ray r = Ray::primary(source_angle, spawn_pixel, PIXEL_EDGE::TOP, edge_dist);

        // Create new ray group for primary ray and add it
        RayGroup primary_ray_group;
        primary_ray_group.my_rays = (Ray*)malloc(max_rays_per_ray_group * sizeof(Ray));
        primary_ray_group.max_size = max_rays_per_ray_group;
        primary_ray_group.my_rays[0] = r; // add the new ray
        primary_ray_group.my_size = 1;

        // Add the new ray group to the appropriate region
        Region region = get_region(spawn_pixel, N, M);
        int region_index = region.first + region.second * L; // index of region within vector of region groups
        region_groups[region_index].push_back(primary_ray_group);
    }
    return;
}

__device__ void spawn_secondary_rays(RayGroup* group, Pixel spawn_pixel, double total_energy, int N)
{
    for (int i = 0; i < PARAM_KS; i++)
    {
        double source_angle = random_source_angle_uniform(); // uniform random source angle
        double partial_energy = total_energy / PARAM_KS;
        Ray new_ray = Ray::secondary_from_center(source_angle, spawn_pixel, partial_energy);
        Pixel current_pixel = new_ray.get_current_pixel();
        if (out_of_bounds(current_pixel, N))
        {
            continue;
        }
        group->my_rays[group->my_size] = new_ray;
        group->my_size++;
    }
    return;
}

////////// THREAD GROUP EVOLUTION //////////

__device__ bool random_interact(Pixel target_pixel, double distance, double* densities, int N)
{
    int i = target_pixel.first, j = target_pixel.second;
    double density = densities[i + j * N];
    double l_ep = density * distance; // effective path length travelled in pixel
    double probability = std::exp(-PARAM_A / l_ep);

    hiprandState state;
    init_curand_state(&state);
    double rand = hiprand_uniform_double(&state);
    return (rand < probability);
}

__device__ void
transfer_energy(Ray* ray, Pixel target_pixel, double unscaled_energy, double* densities, double* doses, int N)
{
    int i = target_pixel.first, j = target_pixel.second;
    double density = densities[i + j * N];
    double energy_to_transfer = unscaled_energy * density; // scale energy by pixel density
    double current_ray_energy = ray->get_current_energy();

    // Ray cannot transfer more energy that it has
    energy_to_transfer = fmin(energy_to_transfer, current_ray_energy);

    // Remove energy from ray and add it to pixel dose
    ray->set_current_energy(current_ray_energy - energy_to_transfer);

    doses[i + j * N] += energy_to_transfer;

    return;
}

// TODO needs to take pointer to a RegroupBuffer as argument
__device__ int evolve_rays(RayGroup* group, int region_index, double* densities, double* doses, int N, int M, RegroupBuffer* g_buffer)
{
    int rays_evolved = 0;
    int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = 0; i < group->my_size; i++)
    {
        Ray* r = &group->my_rays[i];
        // Only evolve active rays
        if (r->is_active())
        {
            // Trace ray
            TraceHistory rtrace = r->trace();
            Pixel visited_pixel = rtrace.visited;
            double travel_distance = rtrace.distance; // distance traveled in visited pixel
            rays_evolved++;
            // printf("block %d, thread %d: ray visited pixel %d,%d\n", blockIdx.x, threadIdx.x, visited_pixel.first,
            //       visited_pixel.second);
            if (r->is_primary()) // primary ray
            {
                if (random_interact(visited_pixel, travel_distance, densities, N))
                {
                    double energy_to_deposit = PARAM_F * travel_distance * r->get_current_energy();
                    transfer_energy(r, visited_pixel, energy_to_deposit, densities, doses, N);
                    spawn_secondary_rays(group, visited_pixel, r->get_current_energy(), N);
                    r->set_current_energy(0);
                }
            }
            else // secondary ray
            {
                double energy_to_deposit = PARAM_G * travel_distance;
                transfer_energy(r, visited_pixel, energy_to_deposit, densities, doses, N);
            }

            // Deactivate ray if out of energy or outside of the grid bounds
            if (r->get_current_energy() < PARAM_EPSILON || out_of_bounds(r->get_current_pixel(), N))
            {
                r->deactivate();
            }

            // Check if the ray is still in the region
            int new_region_index = get_region_index(r->get_current_pixel(), N, M);
            if (new_region_index != region_index)
            {
                r->deactivate();

                // TODO add it to buffer (calling it g_buff, a ptr to  RegroupBuffer)
                int buffer_index = thread_index * g_buffer->section_size +
                                 g_buffer->ray_counts[thread_index];     // this thread's next index in buffer
                g_buffer->rays[buffer_index] = *r;                          // add ray to buffer
                g_buffer->region_indices[buffer_index] = new_region_index; // add destination region index to buffer
                g_buffer->ray_counts[thread_index]++;                    // update buffer size
                
            }
        }
    }
    return rays_evolved;
}

__device__ void evolve_to_completion(RayGroup* group, int region_index, double* densities, double* doses, int N, int M, RegroupBuffer* g_buffer)
{
    int rays_evolved = group->my_size;
    while (rays_evolved > 0)
    {
        rays_evolved = evolve_rays(group, region_index, densities, doses, N, M, g_buffer);
    }
    return;
}

__global__ void run_rays(RayGroup* region_group_arr,
                         int region_group_arr_size,
                         int region_index,
                         double* densities,
                         double* doses,
                         int N,
                         int M,
                         RegroupBuffer* g_buffer)
{
    // printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
    int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_index < region_group_arr_size)
    {
        RayGroup* cur_ray_group = &region_group_arr[thread_index];
        printf("thread index %d: evolving %d rays\n", thread_index, cur_ray_group->my_size);
        evolve_to_completion(cur_ray_group, region_index, densities, doses, N, M, g_buffer);
    }

    return;
}

////////// REGION GROUP RUNNING AND PROCESSING //////////

// add rays from g_buffer to new regions
// 
__host__ void regroup(std::vector<RegionGroup>& region_groups, RegroupBuffer* g_buffer, int max_num_rays, int num_ray_groups)
{
    for (int i = 0; i < num_ray_groups; i++)
    {
        int buffer_index = i * g_buffer->section_size; // index into single ray groups data
        Ray* rays = &g_buffer->rays[buffer_index];      // array of rays to be regrouped
        int *region_indices = &g_buffer->region_indices[buffer_index]; // array of regions corresponding to array of rays
        int num_rays = g_buffer->ray_counts[i];                     // number of rays to be regrouped

        for (int j = 0; j < num_rays; j++)
        {
            Ray cur_ray = rays[j]; // current ray to be regrouped
            cur_ray.reactivate();
            int new_region = region_indices[j]; // region ray is entering
            int region_group_size = region_groups[new_region].size(); // size of the region's last ray group
            RayGroup last_ray_group = region_groups[new_region][region_group_size - 1]; // last ray group in the region group
            
            
            int ray_group_size = last_ray_group.my_size;
            if (ray_group_size < max_num_rays) // if last ray group is not full, add ray to ray group
            {
                last_ray_group.my_rays[ray_group_size] = cur_ray;
                last_ray_group.my_size++;
                region_groups[new_region][region_group_size - 1] = last_ray_group;
            }
            else                             // else add ray to new ray group
            {
                RayGroup new_group;
                new_group.my_rays = (Ray *) malloc(max_num_rays * sizeof(Ray));
                new_group.my_rays[0] = cur_ray;
                new_group.my_size = 1;
                new_group.max_size = max_num_rays;
                region_groups[new_region].push_back(new_group);
            }
            
        }
    }

}

// allocate a regroup buffer on device
__host__ void init_regroup_buffer_cuda(RegroupBuffer* g_buffer, int max_num_rays, int num_ray_groups)
{
    hipMalloc((void **) g_buffer, sizeof(RegroupBuffer));
    g_buffer->section_size = max_num_rays;
    hipMalloc((void **) g_buffer->rays, num_ray_groups * max_num_rays * sizeof(Ray));
    hipMalloc((void **) g_buffer->region_indices, num_ray_groups * max_num_rays * sizeof(int));
    hipMalloc((void **) g_buffer->ray_counts, num_ray_groups * sizeof(int));
}

// allocate a regroup buffer on host and copy the contents of device's regroup buffer to it
__host__ void init_regroup_buffer(RegroupBuffer* g_buffer, RegroupBuffer *g_buffer_cuda, int max_num_rays, int num_ray_groups)
{
    g_buffer = (RegroupBuffer*) malloc(sizeof(RegroupBuffer));
    g_buffer->rays = (Ray*) malloc(num_ray_groups * max_num_rays * sizeof(Ray));
    g_buffer->region_indices = (int*) malloc(num_ray_groups * max_num_rays * sizeof(int));
    g_buffer->ray_counts = (int *) malloc(num_ray_groups * sizeof(int));
    
    hipMemcpy(g_buffer->rays, g_buffer_cuda->rays, num_ray_groups * max_num_rays * sizeof(Ray), hipMemcpyDeviceToHost);
    hipMemcpy(g_buffer->region_indices, g_buffer_cuda->region_indices, num_ray_groups * max_num_rays * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(g_buffer->ray_counts, g_buffer_cuda->ray_counts, num_ray_groups * sizeof(int), hipMemcpyDeviceToHost);
}

__host__ std::vector<int> get_forward_schedule(int L)
{
    std::vector<int> schedule;
    for (int a = 0; a < (2 * L - 1); a++)
    {
        int i = min(a, L - 1);
        int j = max(0, a + 1 - L);
        while (i >= 0 && j < L)
        {
            int task_index = i + j * L;
            i--;
            j++;
            schedule.push_back(task_index);
        }
    }
    return schedule;
}

__host__ void run_region_groups(std::vector<RegionGroup>& region_groups, double* densities, double* doses, int N, int M)
{
    int L = N / M;                                       // number of regions per side
    std::vector<int> schedule = get_forward_schedule(L); // linear indices of regions in diagonal order

    int rays_remaining = 1;
    while (rays_remaining > 0) // TODO: add multiple passes, checking for when all rays are done
    {
        // Forward pass
        for (std::vector<int>::iterator f_it = schedule.begin(); f_it != schedule.end(); f_it++)
        {
            int region_index = *f_it;
            DEBUG(DB_HOST, std::cout << "Forward pass. Running region group " << region_index << std::endl);
            DEBUG(DB_HOST, std::cout << "It has " << region_groups[region_index].size() << " ray groups" << std::endl);
            
            RegionGroup cur_region_group = region_groups[region_index]; // current region group
            RegroupBuffer* g_buffer; // empty host regroup buffer. Will be filled in by run_region_group
            
            run_region_group(cur_region_group, region_index, densities, doses, N, M, g_buffer);
            // TODO get regroup buffer from run_region_group, pass it and region_groups to a new function to do regrouping
            // Remember rays need to be reactivated during regrouping
            regroup(region_groups, g_buffer, cur_region_group[0].max_size, cur_region_group.size());
            free(g_buffer);
        }

        // Reverse pass
        for (std::vector<int>::reverse_iterator r_it = schedule.rbegin(); r_it != schedule.rend(); r_it++)
        {
            int region_index = *r_it;
            DEBUG(DB_HOST, std::cout << "Reverse pass. Running region group " << region_index << std::endl);
            DEBUG(DB_HOST, std::cout << "It has " << region_groups[region_index].size() << " ray groups" << std::endl);
            
            RegionGroup cur_region_group = region_groups[region_index];
            RegroupBuffer* g_buffer;
            
            run_region_group(cur_region_group, region_index, densities, doses, N, M, g_buffer);
            regroup(region_groups, g_buffer, cur_region_group[0].max_size, cur_region_group.size());
            free(g_buffer);
        }

        rays_remaining = 0;
    }
    return;
}

// TODO I think this should return a host regroup buffer that can be passed to another function to handle the actual
// regrouping
// JA: I didn't return it, just passed and modified the pointer to host regroup buffer within this function
__host__ void
run_region_group(RegionGroup& region_group, int region_index, double* densities, double* doses, int N, int M, RegroupBuffer* g_buffer)
{
    // Set device memory limits
    size_t heap_limit = 1 << 26; // default 8MB, this sets to 64 MB
    hipDeviceSetLimit(hipLimitMallocHeapSize, heap_limit);

    // First copy rays to ray groups on device, done by just replacing host pointers with device pointers
    int num_ray_groups = region_group.size(); // number of ray groups in current region group
    DEBUG(DB_HOST, std::cout << "Copying rays from host to device" << std::endl); 
    int max_num_rays = region_group[0].max_size; // all ray groups have same max size so just get any max size
    for (int g = 0; g < num_ray_groups; g++)
    {
        
        Ray* rays_cuda;
        hipMalloc(&rays_cuda, max_num_rays * sizeof(Ray)); // allocated memory on device
        hipMemcpy(rays_cuda, region_group[g].my_rays, max_num_rays * sizeof(Ray),
                   hipMemcpyHostToDevice);               // copy from host to device
        Ray* old_host_rays_ptr = region_group[g].my_rays; // pointer to rays on host
        region_group[g].my_rays = rays_cuda;              // this is now a device pointer NOT a host pointer
        free(old_host_rays_ptr);                          // free host memory
    }

    DEBUG(DB_HOST, std::cout << "Copying ray groups from host to device" << std::endl);
    // Copy region group to GPU (std::vector on host to array on device)
    RayGroup* region_group_cuda_arr;
    hipMalloc(&region_group_cuda_arr, num_ray_groups * sizeof(RayGroup)); // allocated memory on device
    hipMemcpy(region_group_cuda_arr, region_group.data(), num_ray_groups * sizeof(RayGroup),
               hipMemcpyHostToDevice); // copy from host to device

    // Clear region group vector because we messed with its memory, and its rays are all going to be run
    region_group.clear();

    // TODO allocate g_buffer on DEVICE
    // We need to be a little careful in making sure the section size of the buffer is always enough to handle 
    // all rays from a given thread group. We could check in the above for loop for the largest "max_num_rays" 
    // found and use that (even though we will probably make all the ray groups with the same max_size)
    
    // way of having host and device share memory. Might be useful later
    /*
    int *flags;
    int flag_size = 1;
    hipHostMalloc((void**) &flags, flag_size * sizeof(int));
    memset(flags, 0, flag_size * sizeof(int));
    */ 
    RegroupBuffer* g_buffer_cuda; // empty device regroup buffer
    init_regroup_buffer_cuda(g_buffer_cuda, max_num_rays, num_ray_groups); // allocate device regroup buffer
    
    // TODO make sure there are enough threads to handle all ray groups
    // Run thread groups in parallel
    int grid_size = num_ray_groups / 1024; // 1024 is max threads in a block
    int block_size = num_ray_groups % 1024;
    run_rays<<<grid_size, block_size>>>(region_group_cuda_arr, num_ray_groups, region_index, densities, doses, N, M, g_buffer_cuda);
    
    // Wait for GPU computation to finish
    hipDeviceSynchronize();
    
    // Free device memory
    // TODO free ray group pointers
    hipFree(region_group_cuda_arr); 

    // TODO copy g_buffer back to host buffer
    init_regroup_buffer(g_buffer, g_buffer_cuda, max_num_rays, num_ray_groups); // copy g_buffer back to host buffer
    hipFree(g_buffer_cuda);

    return;
}

int main(void)
{
    int N = 1000;                 // grid size in pixels per side
    int M = 100;                  // region size in pixels per side
    int num_primary_rays = 10000; // number of primary rays to run

    // NOTE: all 2D arrays unwrapped as 1D arrays/vectors use linear indexing
    // of the form i,j -> i + j * edge_dimension

    // Set up grids
    DEBUG(DB_HOST, std::cout << "Starting simulation, allocating grids" << std::endl);
    double *densities, *doses;
    hipMallocManaged(&densities, N * N * sizeof(double));
    hipMallocManaged(&doses, N * N * sizeof(double));

    // Initialize densities and doses, write densities
    DEBUG(DB_HOST, std::cout << "Initializing densities" << std::endl);
    initialize_densities_constant(densities, N, 0.3);
    DEBUG(DB_HOST, std::cout << "Initializing doses to zero" << std::endl);
    initialize_doses(doses, N);
    DEBUG(DB_HOST, std::cout << "Writing densities" << std::endl);
    write_to_csv_file(densities, N, "densities.csv");

    // Set up region groups
    int L = N / M;           // number of regions per side
    int num_regions = L * L; // total number of regions
    DEBUG(DB_HOST, std::cout << "Number of regions: " << num_regions << std::endl);
    int max_rays_per_ray_group = PARAM_KS + 1;           // initial capacity of a ray group
    std::vector<RegionGroup> region_groups(num_regions); // vector of region groups

    DEBUG(DB_HOST, std::cout << "Spawning primary rays to region groups" << std::endl);
    // Spawn primary rays to region groups (vector is passed by reference)
    spawn_primary_rays(region_groups, num_primary_rays, max_rays_per_ray_group, N, M);
    
    DEBUG(DB_HOST, std::cout << "Running region groups" << std::endl);
    // Run region groups until complete
    run_region_groups(region_groups, densities, doses, N, M);

    // Write out doses
    DEBUG(DB_HOST, std::cout << "Writing doses" << std::endl);
    write_to_csv_file(doses, N, "doses.csv");

    // Free memory
    hipFree(densities);
    hipFree(doses);

    DEBUG(DB_HOST, std::cout << "All done" << std::endl);
    return 0;
}

