#include "hip/hip_runtime.h"
#include "../parameters.hpp"
#include "base-simulation.cuh"

#include <chrono>
#include <iostream>
#include <stdlib.h>

int main(int argc, char** argv)
{
    if (argc < 4)
    {
        std::cerr << "Invalid number of arguments [grid size] [number of primary rays] [density "
                     "initialization method] "
                     "[additional]"
                  << std::endl;
        std::cerr << "Initialization methods and their additional arguments are:" << std::endl;
        std::cerr << "C - Constant [density]" << std::endl;
        std::cerr << "R - Random" << std::endl;
        std::cerr << "G - Gaussian [maximum density] [spread]" << std::endl;
        std::cerr << "M - Multiple random Gaussians [number of Gaussians] [maximum density] [spread]" << std::endl;
        exit(1);
    }

    // Grid size
    const int N = atoi(argv[1]);
    if (N < 1)
    {
        std::cerr << "Grid size must be greater than 0" << std::endl;
        exit(1);
    }

    // Number of rays
    const int num_primary_rays = atoi(argv[2]);
    if (num_primary_rays < 1)
    {
        std::cerr << "Ray count must be greater than 0" << std::endl;
        exit(1);
    }

    // NOTE: all 2D arrays unwrapped as 1D arrays/vectors use linear indexing
    // of the form i,j -> i + j * edge_dimension

    // Set up grids
    DEBUG(DB_HOST, std::cout << "Starting simulation, allocating grids" << std::endl);
    double *densities, *doses;
    hipMallocManaged(&densities, N * N * sizeof(double));
    hipMallocManaged(&doses, N * N * sizeof(double));

    // Initialize densities and doses
    DEBUG(DB_HOST, std::cout << "Initializing doses to zero" << std::endl);
    initialize_doses(doses, N);
    DEBUG(DB_HOST, std::cout << "Initializing densities" << std::endl);

    // Initialize densities according to given method
    int init_method_arg = 3;
    switch (argv[init_method_arg][0])
    {
    case 'C': // constant
    {
        if (argc < (init_method_arg + 1 + 1))
        {
            std::cerr << "Not enough arguments for chosen initialization option" << std::endl;
            exit(1);
        }

        const double density = strtod(argv[init_method_arg + 1], NULL);
        if (density < 0 || density > 1)
        {
            std::cerr << "Density must be between 0 and 1" << std::endl;
            exit(1);
        }
        initialize_densities_constant(densities, N, density);
        break;
    }
    case 'R': // random
    {
        initialize_densities_random(densities, N);
        break;
    }

    case 'G': // centered Gaussian
    {
        if (argc < (init_method_arg + 1 + 2))
        {
            std::cerr << "Not enough arguments for chosen initialization option" << std::endl;
            exit(1);
        }

        const double max_density = strtod(argv[init_method_arg + 1], NULL);
        if (max_density < 0 || max_density > 1)
        {
            std::cerr << "Density must be between 0 and 1" << std::endl;
            exit(1);
        }
        const double spread = strtod(argv[init_method_arg + 2], NULL);

        initialize_densities_centered_gaussian(densities, N, max_density, spread);
        break;
    }

    case 'M': // multiple random Gaussians
    {
        if (argc < (init_method_arg + 1 + 3))
        {
            std::cerr << "Not enough arguments for chosen initialization option" << std::endl;
            exit(1);
        }

        const int number = atoi(argv[init_method_arg + 1]);
        if (number < 1)
        {
            std::cerr << "Number of Gaussians must be greater than 0" << std::endl;
            exit(1);
        }

        const double max_density = strtod(argv[init_method_arg + 2], NULL);
        if (max_density < 0 || max_density > 1)
        {
            std::cerr << "Density must be between 0 and 1" << std::endl;
            exit(1);
        }
        const double spread = strtod(argv[init_method_arg + 3], NULL);

        initialize_densities_random_gaussians(densities, N, number, max_density, spread);
        break;
    }

    default:
    {
        std::cerr << "Invalid initialization method given" << std::endl;
        exit(1);
        break;
    }
    }

    DEBUG(DB_GENERAL, std::cout << "Densities initialized" << std::endl << std::endl);

    // Write densities
    std::cout << "Writing densities" << std::endl;
    write_to_csv_file(densities, N, "densities.csv");

    // Set up simulation 
    hipDeviceSetLimit(hipLimitMallocHeapSize, GPU_HEAP_LIMIT); 
    int grid_size = max(1, num_primary_rays / GPU_BLOCK_SIZE); // number of thread blocks
    int block_size = GPU_BLOCK_SIZE;                             

    // TIMER START
    auto start = std::chrono::high_resolution_clock::now(); 

    std::cout << "Spawning and running " << num_primary_rays << " primary rays" << std::endl;    
    // Run rays on threads (includes spawning)
    run_rays<<<grid_size, block_size>>>(num_primary_rays, densities, doses, N);

    // Wait for GPU computation to finish
    hipDeviceSynchronize();

    // TIMER END
    auto finish = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = finish - start;
    std::cout << "Elapsed time: " << elapsed.count() << " s" << std::endl;

    // Write result
    std::cout << "Writing doses" << std::endl;
    write_to_csv_file(doses, N, "doses.csv");

    // Free memory
    hipFree(densities);
    hipFree(doses);

    std::cout << "All done" << std::endl;
    return 0;
}

