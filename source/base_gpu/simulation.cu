#include "hip/hip_runtime.h"
#include "simulation.cuh"
#include <stdio.h>

__host__ void initialize_doses(double* doses, int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            doses[i * N + j] = 0;
        }
    }
    return;
}

__host__ void initialize_densities_random(double* densities, int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            densities[i * N + j] = uniform_dist(random_engine);
        }
    }
    return;
}

__host__ void initialize_densities_constant(double* densities, int N, double density)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            densities[i * N + j] = density;
        }
    }
    return;
}

__host__ void initialize_densities_centered_gaussian(double* densities, int N, double max_density, double spread)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            int mid = N / 2; // middle pixel
            int x = i - mid;
            int y = j - mid;
            double std_dev = spread * N;
            densities[i * N + j] = max_density * exp(-(x * x + y * y) / (2 * std_dev * std_dev));
        }
    }
    return;
}

__host__ void
initialize_densities_random_gaussians(double* densities, int N, int n_gaussians, double max_density, double spread)
{
    double std_dev = spread * N;
    double highest = 0;

    // Add the Gaussians
    for (int k = 0; k < n_gaussians; k++)
    {
        int mid_x = floor(uniform_dist(random_engine) * N);
        int mid_y = floor(uniform_dist(random_engine) * N);
        for (int i = 0; i < N; i++)
        {
            for (int j = 0; j < N; j++)
            {
                int x = i - mid_x;
                int y = j - mid_y;
                densities[i * N + j] += max_density * exp(-(x * x + y * y) / (2 * std_dev * std_dev));
                highest = fmax(highest, densities[i * N + j]);
            }
        }
    }

    // Normalize the resulting density distribution
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            densities[i * N + j] = max_density * densities[i * N + j] / highest;
        }
    }
    return;
}

__host__ void write_to_csv_file(double* grid_data, int N, const std::string& filename)
{
    std::ofstream output;
    output.open(filename);
    for (int j = 0; j < N; j++)
    {
        for (int i = 0; i < N - 1; i++)
        {
            output << grid_data[i * N + j] << ",";
        }
        output << grid_data[N - 1 + j] << "\n";
    }
    output.close();
    return;
}

__device__ void init_curand_state(hiprandState_t* state)
{
    // Initialize random kernel
    int tId = threadIdx.x + (blockIdx.x * blockDim.x);
    hiprand_init((unsigned long long)clock(), tId, 0, state);
    //printf("myID: %d\n", tId);
    return;
}

__device__ double uniform_angle_dist(hiprandState_t* state) { return 2 * M_PI * hiprand_uniform_double(state); }

__device__ double normal_dist(hiprandState_t* state, double mean, double std_dev)
{
    return mean + std_dev * hiprand_normal_double(state);
}
__device__ double random_source_angle(bool normal)
{
    hiprandState state;
    init_curand_state(&state);

    double angle;
    if (normal)
    { // Normal distribution
        angle = normal_dist(&state, PARAM_MEAN, PARAM_SIGMA);
    }
    else
    { // Uniform distribution between 0 and 2 pi
        angle = uniform_angle_dist(&state);
    }

    // Normalize angle
    while (angle < 0.0)
    {
        angle += 2 * M_PI;
    }
    while (angle >= 2 * M_PI)
    {
        angle -= 2 * M_PI;
    }
    return angle;
}

__device__ bool out_of_bounds(Pixel current_pixel, int N)
{
    return (current_pixel.first < 0 || current_pixel.first >= N || current_pixel.second < 0 ||
            current_pixel.second >= N);
}

__device__ void spawn_primary_ray(RayGroup* group, int N)
{
    // Randomly select source angle from normal distribution
    double source_angle = random_source_angle(true);

    printf("block %d, thread %d spawning primary ray with angle %0.5f\n", blockIdx.x, threadIdx.x, source_angle);

    // Calculate initial ray position
    double horiz_dist_from_center = PARAM_D * N * tan(source_angle); // horizontal distance from center of top edge
    int middle_pixel = N / 2;
    double horiz_dist_from_left = middle_pixel + horiz_dist_from_center;

    // If ray does not miss grid entirely, spawn it
    if (horiz_dist_from_left < 0 || horiz_dist_from_left >= N ||
        (source_angle >= M_PI / 2 && source_angle <= 3 * M_PI / 2))
    {
        // DEBUG(DB_INIT_PRI, std::cout << "New primary ray missed the grid, not adding" << std::endl);
    }
    else
    {
        double horiz_dist_from_left_rounded = floor(horiz_dist_from_left);
        Pixel spawn_pixel;
        spawn_pixel.first = horiz_dist_from_left_rounded;
        spawn_pixel.second = 0; // always starts from top of grid
        double edge_dist = horiz_dist_from_left - horiz_dist_from_left_rounded;
        group->my_rays[group->my_size] = Ray::primary(source_angle, spawn_pixel, PIXEL_EDGE::TOP, edge_dist);
        group->my_size++;
        // DEBUG(DB_INIT_PRI, std::cout << "New primary ray added at pixel " << pixel.first << "," << pixel.second
        //                                     << " with angle " << source_angle << std::endl);
    }
    return;
}

__device__ void spawn_secondary_rays(RayGroup* group, Pixel spawn_pixel, double total_energy, int N)
{
    // DEBUG(DB_INIT_SEC, std::cout << "Spawning " << PARAM_KS << " secondary rays from pixel " << spawn_pixel.first <<
    // ","
    //                                 << spawn_pixel.second << "..." << std::endl);
    for (int i = 0; i < PARAM_KS; i++)
    {
        double source_angle = random_source_angle(false); // uniform random source angle
        double partial_energy = total_energy / PARAM_KS;
        Ray new_ray = Ray::secondary_from_center(source_angle, spawn_pixel, partial_energy);
        Pixel current_pixel = new_ray.get_current_pixel();
        if (out_of_bounds(current_pixel, N))
        {
            // DEBUG(DB_INIT_SEC, std::cout << "Ray is out of bounds, not adding" << std::endl);
        }
        else
        {
            group->my_rays[group->my_size] = new_ray;
            group->my_size++;

            // DEBUG(DB_INIT_SEC, std::cout << "Ray is in bounds, adding" << std::endl);
        }
    }
    // DEBUG(DB_INIT_SEC, std::cout << "Done" << std::endl << std::endl);
    return;
}

__device__ bool random_interact(Pixel target_pixel, double distance, double* densities, int N)
{
    int i = target_pixel.first, j = target_pixel.second;
    double density = densities[i * N + j];
    double l_ep = density * distance; // effective path length travelled in pixel
    double probability = std::exp(-PARAM_A / l_ep);

    hiprandState state;
    init_curand_state(&state);
    double rand = hiprand_uniform_double(&state);
    return (rand < probability);
}

__device__ void
transfer_energy(Ray* ray, Pixel target_pixel, double unscaled_energy, double* densities, double* doses, int N)
{
    int i = target_pixel.first, j = target_pixel.second;
    double density = densities[i * N + j];
    double energy_to_transfer = unscaled_energy * density; // scale energy by pixel density
    double current_ray_energy = ray->get_current_energy();

    // Ray cannot transfer more energy that it has
    energy_to_transfer = fmin(energy_to_transfer, current_ray_energy);

    // Remove energy from ray and add it to pixel dose
    ray->set_current_energy(current_ray_energy - energy_to_transfer);
    //printf("Attempting to transfer %0.6f energy\n", energy_to_transfer);
    //printf("Pixel energy before transfer: %0.6f\n", doses[i * N + j]);
    doses[i * N + j] += energy_to_transfer;
    //printf("Pixel energy after transfer: %0.6f\n", doses[i * N + j]);
    return;
}

__device__ int evolve_rays(RayGroup* group, double* densities, double* doses, int N)
{
    int rays_evolved = 0;

    for (int i = 0; i < group->my_size; i++)
    {
        Ray* r = &group->my_rays[i];
        // Only evolve active rays
        if (r->is_active())
        {
            // Trace ray
            // DEBUG(DB_TRACE, std::cout << "Tracing ray " << i << std::endl);
            TraceHistory rtrace = r->trace();
            Pixel visited_pixel = rtrace.visited;
            double travel_distance = rtrace.distance; // distance traveled in visited pixel
            rays_evolved++;

            if (r->is_primary()) // primary ray
            {
                // DEBUG(DBevolve_PRI, std::cout << "Primary ray " << i << "  visited pixel " << visited_pixel.first
                //                                               << "," << visited_pixel.second << " with travel dist "
                //                                               << travel_distance
                //                                               << std::endl);
                if (random_interact(visited_pixel, travel_distance, densities, N))
                {
                    printf("block %d, thread %d primary ray interacted at pixel %d,%d\n", blockIdx.x, threadIdx.x,
                           visited_pixel.first, visited_pixel.second);
                    // DEBUG(DBevolve_PRI, std::cout << "Primary ray " << i << " interacted" << std::endl);
                    // Deposit energy to pixel
                    // DEBUG(DBevolve_PRI, std::cout << "Depositing energy to pixel" << std::endl);
                    // DEBUG(DBevolve_PRI, std::cout << "Starting energy " << r->get_current_energy() << std::endl);
                    printf("distance traveled %0.6f\n", travel_distance);
                    double energy_to_deposit = PARAM_F * travel_distance * r->get_current_energy();
                    transfer_energy(r, visited_pixel, energy_to_deposit, densities, doses, N);
                    // DEBUG(DBevolve_PRI, std::cout << "Energy after deposit " << r->get_current_energy() <<
                    // std::endl); DEBUG(DBevolve_PRI, std::cout << "Will spawn secondary rays next" << std::endl <<
                    // std::endl); Spawn secondary rays, transferring remaining energy to them
                    spawn_secondary_rays(group, visited_pixel, r->get_current_energy(), N);
                    r->set_current_energy(0);
                }
                else
                {
                    // DEBUG(DBevolve_PRI, std::cout << "No interaction" << std::endl << std::endl);
                }
            }
            else // secondary ray
            {
                // DEBUG(DBevolve_SEC, std::cout << "Secondary ray " << i << " visited pixel " << visited_pixel.first
                //                                               << "," << visited_pixel.second << " with travel dist "
                //                                               << travel_distance
                //                                               << std::endl);
                double energy_to_deposit = PARAM_G * travel_distance;
                // DEBUG(DBevolve_SEC, std::cout << "Depositing energy to pixel" << std::endl);
                // DEBUG(DBevolve_SEC, std::cout << "Starting energy " << r->get_current_energy() << std::endl);
                // DEBUG(DBevolve_SEC, std::cout << "Unscaled energy to deposit " << energy_to_deposit << std::endl);
                transfer_energy(r, visited_pixel, energy_to_deposit, densities, doses, N);
                // DEBUG(DBevolve_SEC, std::cout << "Energy after deposit " << r->get_current_energy() << std::endl
                //                                               << std::endl);
            }

            // Deactivate ray if out of energy or outside of the grid bounds
            if (r->get_current_energy() < PARAM_MINERGY || out_of_bounds(r->get_current_pixel(), N))
            {
                // DEBUG(DBevolve_SEC, std::cout << "Ray " << i << " is out of energy or bounds, deactivating"
                //                                               << std::endl
                //                                               << std::endl);
                r->deactivate();
            }
        }
    }
    return rays_evolved;
}

__device__ void evolve_to_completion(RayGroup* group, double* densities, double* doses, int N)
{
    int rays_evolved = group->my_size;
    //  int raysevolved = ray_group->size();
    while (rays_evolved > 0)
    {
        rays_evolved = evolve_rays(group, densities, doses, N);
        // DEBUG(DB_GENERAL, std::cout << raysevolved << " rays evolved" << std::endl);
        // DEBUG(DB_GENERAL, std::cout << (ray_group->size() - prev_num_rays) << " rays added" << std::endl <<
        // std::endl)
    }

    return;
}
__device__ void run_serial(int num_primary_rays, double* densities, double* doses, int N)
{
    // Each primary ray is done serially as its own individual ray group
    for (int i = 0; i < num_primary_rays; i++)
    {
        printf("Hello from block %d, thread %d. I'm running primary ray %d\n", blockIdx.x, threadIdx.x, i);
        // Just running one primary ray (and its secondaries) at a time for now
        // TODO: might want to change how memory allocation/reallocation is done
        RayGroup primary_ray_group;
        int max_num_rays = 1 + PARAM_KS; // at most one primary ray plus all secondaries
        Ray* rays;
        rays = (Ray*)malloc(max_num_rays * sizeof(Ray));
        primary_ray_group.my_rays = rays;
        primary_ray_group.my_size = 0;

        // DEBUG(DB_INIT_PRI, std::cout << "Spawning " << num_primary_rays << " primary rays..." << std::endl);
        spawn_primary_ray(&primary_ray_group, N);
        // DEBUG(DB_INIT_PRI, std::cout << "Done, " << ray_group.size() << " rays added" << std::endl << std::endl);
        // DEBUG(DB_GENERAL, std::cout << "Evolving rays..." << std::endl)
        evolve_to_completion(&primary_ray_group, densities, doses, N);
        // DEBUG(DB_GENERAL, std::cout << "Done" << std::endl);
        free(primary_ray_group.my_rays);
    }
    return;
}

/// Kernel function for base-GPU
__global__ void run_rays(int num_primary_rays, double* densities, double* doses, int N)
{
    run_serial(num_primary_rays, densities, doses, N);
}

// TODO could actually make N a command line argument, right?
int main(void)
{
    DEBUG(DB_GPU, std::cout << "Starting simulation, allocating grids" << std::endl);
    int N = 10000; // grid size in pixels per side

    // Storing the N by N grid data as 1D arrays of length N*N
    // such that element i,j is at index i * N + j
    // Currently for the model I think i corresponds to x and j corresponds to y
    double *densities, *doses;
    hipMallocManaged(&densities, N * N * sizeof(double));
    hipMallocManaged(&doses, N * N * sizeof(double));

    DEBUG(DB_GPU, std::cout << "Initializing densities" << std::endl);
    // initialize_densities_random(densities, N);
    initialize_densities_constant(densities, N, 0.3);

    DEBUG(DB_GPU, std::cout << "Initializing doses to zero" << std::endl);
    initialize_doses(doses, N);
    size_t heap_limit = pow(2, 26); // default 8MB, this sets to 32 MB
    hipDeviceSetLimit(hipLimitMallocHeapSize, heap_limit);
 
    DEBUG(DB_GPU, std::cout << "Writing densities" << std::endl);
    //write_to_csv_file(densities, N, "../../plot/densities.csv");
    write_to_csv_file(densities, N, "densities.csv");
    
    int grid_size = 256; // number of thread blocks
    int block_size = 128;   // TODO 1 thread per block, does this make sense?

    DEBUG(DB_GPU, std::cout << "Running rays on threads" << std::endl);
    int primary_rays_per_thread = 32; // each thread does this many rays in serial
    run_rays<<<grid_size, block_size>>>(primary_rays_per_thread, densities, doses, N);

    // Wait for GPU computation to finish
    hipDeviceSynchronize();

    DEBUG(DB_GPU, std::cout << "Writing doses" << std::endl);
    //write_to_csv_file(doses, N, "../../plot/doses.csv");
    write_to_csv_file(doses, N, "doses.csv");
    DEBUG(DB_GPU, std::cout << "I'm the main function, look at me!" << std::endl);

    hipFree(densities);
    hipFree(doses);

    return 0;
}

